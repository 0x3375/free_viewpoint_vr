#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <chrono>

#define USE_ZERO_COPY

class StopWatch {
public:
	void Start() {
		t0 = std::chrono::high_resolution_clock::now();
	}
	double Stop() {
		return std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch() - t0.time_since_epoch()).count();
	}

private:
	std::chrono::high_resolution_clock::time_point t0;
};


__global__ void ParallelFor(unsigned char* IMG, unsigned char* LF, double Alloc_Angle_s, double times, int Y, int POS_Y, int POS_X, int LFUW, int DATAW, int WIDTH, int HEIGHT, int out_w, int dir)
{
	int tw = blockIdx.x; // blockIdx.x = (int)[0, (out_w - 1)]
	int th = threadIdx.y; // threadIdx = (int)[0, (HEIGHT - 1)]

	//Console.WriteLine("{0}: {1}", Thread.CurrentThread.ManagedThreadId, w);
	double a = Alloc_Angle_s + (0.0025 * (double)tw);
	double P = (double)(Y - POS_Y) * tan(a) + POS_X;
	double b = sqrt(2.0) * LFUW;
	double N_dist = sqrt((double)((P - POS_X) * (P - POS_X) + (Y - POS_Y) * (Y - POS_Y))) / b;

	P = P / 2;
	int P_1 = (int)(round(P + (DATAW / 2)));
	if (dir == 3 || dir == 4) P_1 = DATAW - P_1 - 1;
	
	double U = a * (180.0 / 3.14159265358979323846) * (1.0 / 180.0) * (WIDTH / 2) + (WIDTH / 2);
	int U_1 = (int)(round(U));

	if (dir == 2) U_1 = U_1 + WIDTH / 4;
	if (dir == 3) U_1 = U_1 + WIDTH / 2;
	if (dir == 4) U_1 = U_1 - WIDTH / 4;

	if (U_1 >= (WIDTH)) U_1 = U_1 - WIDTH;
	else if (U_1 < 0) U_1 = U_1 + WIDTH;

	if (P_1 >= DATAW) P_1 = DATAW - 1;
	else if (P_1 < 0) P_1 = 0;

	if (U_1 >= WIDTH) U_1 = WIDTH - 1;
	else if (U_1 < 0) U_1 = 0;

	int N_off = (int)(floor(times * N_dist + 0.5)) >> 1;
	double N_H_r = (double)(HEIGHT + N_off) / HEIGHT;


	double h_n = (th - HEIGHT / 2) * N_H_r + HEIGHT / 2;

	int U_1_n = 0;
	if (h_n < 0)
	{
		U_1_n = U_1 + WIDTH / 2;
		if (U_1_n > WIDTH - 1) U_1_n = U_1 - WIDTH / 2;

		h_n = (-1 * h_n) - 1;
	}
	else if (h_n > HEIGHT - 1)
	{
		U_1_n = U_1 + WIDTH / 2;
		if (U_1_n > WIDTH - 1) U_1_n = U_1 - WIDTH / 2;

		h_n = HEIGHT - ((h_n - HEIGHT) - 1);
	}
	else
	{
		U_1_n = U_1;
	}

	int H_1 = (int)(round(h_n));
	if (H_1 >= HEIGHT) H_1 = HEIGHT - 1;
	else if (H_1 < 0) H_1 = 0;

	unsigned char PEL_0 = LF[(P_1) * (HEIGHT * WIDTH * 3) + U_1 * (HEIGHT * 3) + H_1 * 3 + 0]; // b
	unsigned char PEL_1 = LF[(P_1) * (HEIGHT * WIDTH * 3) + U_1 * (HEIGHT * 3) + H_1 * 3 + 1]; // g
	unsigned char PEL_2 = LF[(P_1) * (HEIGHT * WIDTH * 3) + U_1 * (HEIGHT * 3) + H_1 * 3 + 2]; // r

	IMG[(HEIGHT - th - 1) * (out_w * 3) + tw * 3 + 2] = PEL_0; // r
	IMG[(HEIGHT - th - 1) * (out_w * 3) + tw * 3 + 1] = PEL_1; // g
	IMG[(HEIGHT - th - 1) * (out_w * 3) + tw * 3 + 0] = PEL_2; // b // height�� ���� ä��� �ڵ�
	// Unity use the big endian color format -> 0xRR 0xGG 0xBB ...
	// Unity use the Bottom-Left origin in 2D coor. (h -> HEIGHT - h) (082219)
}

extern "C" __declspec(dllexport) int CudaParallelFor(unsigned char* h_outResult, unsigned char* h_inLightField, double Alloc_Angle_s, double times, int Y, int POS_Y, int POS_X, int LFUW, int DATAW, int WIDTH, int HEIGHT, int out_w, int dir, int inLightField_Len)
{
	// outResult[HEIGHT * out_W * 3]
	// inLightField[

	unsigned char* h_tmp, *d_tmp;

	

	StopWatch sw_inLightFieldMalloc, sw_ResultMalloc, sw_copyH2D, sw_For, sw_copyD2H, sw_freeinLightField, sw_freeResult;
	hipError_t errorCode;

	dim3 threadsPerBlock(1, HEIGHT); // 1x1024 ����� �����尡 �� ����� ����
	dim3 blocksPerGrid(out_w, 1); // �� ��ϵ��� out_w�� �ʺ�ŭ �� �׸��带 ����

	int alignedLen_I = (((int)(inLightField_Len + 4095) / 4096) * 4096) * sizeof(unsigned char);
	int alignedLen_O = (((int)((HEIGHT * out_w * 3) + 4095) / 4096) * 4096) * sizeof(unsigned char);
	
	unsigned char* d_outResult; // output
	unsigned char* d_inLightField;

	hipSetDeviceFlags(hipDeviceMapHost);

	/******************************/
	errorCode = hipHostAlloc((void**)&h_tmp, inLightField_Len, hipHostMallocMapped);
	if (errorCode != hipSuccess) return errorCode;
	hipMemcpy(h_tmp, h_inLightField, inLightField_Len, hipMemcpyHostToHost);
	errorCode = hipHostAlloc((void**)&d_tmp, (HEIGHT * out_w * 3), hipHostMallocMapped);
	if (errorCode != hipSuccess) return errorCode;
	hipMemcpy(d_tmp, h_outResult, (HEIGHT * out_w * 3), hipMemcpyHostToHost);

	ParallelFor << <blocksPerGrid, threadsPerBlock >> > (d_tmp, h_tmp, Alloc_Angle_s, times, Y, POS_Y, POS_X, LFUW, DATAW, WIDTH, HEIGHT, out_w, dir);
	hipMemcpy(h_outResult, d_tmp, (HEIGHT * out_w * 3), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	errorCode = hipGetLastError();
	if (errorCode != hipSuccess) return errorCode;

	hipHostFree(d_tmp);
	hipHostFree(h_tmp);
	/******************************/
	/*
	sw_ResultMalloc.Start();
	errorCode = hipHostRegister(h_outResult, alignedLen_O, hipHostRegisterMapped);
	// errorCode = hipMalloc((void**)&d_outResult, HEIGHT * out_w * 3 * sizeof(unsigned char));
	double us_resultMalloc = sw_ResultMalloc.Stop() / 1000;
	if (errorCode != hipSuccess) return errorCode;

	sw_inLightFieldMalloc.Start();
	errorCode = hipHostRegister(h_inLightField, alignedLen_I, hipHostRegisterMapped);
	double us_inLightFieldMalloc = sw_inLightFieldMalloc.Stop() / 1000;
	if (errorCode != hipSuccess) return errorCode;

	sw_copyH2D.Start();
	errorCode = hipHostGetDevicePointer((void**)&d_outResult, h_outResult, 0);
	errorCode = hipHostGetDevicePointer((void**)&d_inLightField, h_inLightField, 0);
	double us_copyH2D = sw_copyH2D.Stop() / 1000;
	if (errorCode != hipSuccess) return errorCode;

	sw_For.Start();
	ParallelFor << <blocksPerGrid, threadsPerBlock >> > (d_outResult, d_inLightField, Alloc_Angle_s, times, Y, POS_Y, POS_X, LFUW, DATAW, WIDTH, HEIGHT, out_w, dir);
	hipDeviceSynchronize();
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	
	double us_for = sw_For.Stop() / 1000;

	errorCode = hipGetLastError();
	if (errorCode != hipSuccess) return errorCode;

	//sw_copyD2H.Start();
	//errorCode = hipMemcpy(outResult, d_outResult, HEIGHT * out_w * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	//double us_copyD2H = sw_copyD2H.Stop() / 1000;
	//if (errorCode != hipSuccess) return errorCode;

	// sw_freeResult.Start();
	// errorCode = hipFree(d_outResult);
	// double us_freeResult = sw_freeResult.Stop() / 1000;
	// if (errorCode != hipSuccess) return errorCode;

	// sw_freeinLightField.Start();
	// errorCode = hipFree(d_inLightField);
	// double us_freeinLightField = sw_freeinLightField.Stop() / 1000;
	// if (errorCode != hipSuccess) return errorCode;

	hipHostUnregister(h_inLightField);
	hipHostUnregister(h_outResult);

	double us_total = us_for + us_copyH2D + us_inLightFieldMalloc + us_resultMalloc;// +us_freeinLightField + us_freeResult + us_copyD2H;

	// fprintf(fp, "ResultMalloc\tinLightFieldMalloc\tmemCpyH2D\tFor\tmemCpyD2H\tfreeResult\tfreeinLightField\tTOTAL\n");
	FILE* fp = fopen("C:/myUnity/timelog/cuda.txt", "a");
	fprintf(fp, "%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\n", us_resultMalloc, us_inLightFieldMalloc, us_copyH2D, us_for, us_total);//us_freeinLightField, us_freeResult, us_copyD2H);
	fclose(fp);
	*/
	return 0;
}


/*
__global__ void addcuda(int* a, int *b, int* c)
{
	 c[blockIdx.x * 1024 + threadIdx.y] = 1; // [0, 2047] * 1024 + [0, 1023]

	// blockDim : CUDA Block�� ũ��, Threads�� ���� (threadsPerBlock�� ���� ��) 
	// blockIdx : ��� �ε��� ( 0 ~ blocksPerGrid-1 )
	// ThreadsIdx : Threads �ε��� ( 0 ~ threadsPerBlock-1 )
}

__host__ void cudatest(int a, int b, int c)
{
	int WID = 2048; int HEI = 1024;
	dim3 threadsPerBlock(1, 1024); // # of threads, MAX : X * Y <= 1024 // ��ϴ� 1024�� �����带 ����
	dim3 blocksPerGrid(WID / threadsPerBlock.x, HEI / threadsPerBlock.y); // # ����� ��� ����Ǿ�� �ϴ°�? = �� �׸���� �� ������� �����Ǵ°�?)

	int* d_a;
	int* d_b;
	int* d_c;

	int* arr = (int*)malloc(sizeof(int) * 2048 * 1024);
	
	memset(arr, 0, sizeof(int) * 2048 * 1024);
	

	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc((void**)&d_b, sizeof(int));

	hipMalloc((void**)&d_c, 2048 * 1024 * sizeof(int));


	
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, arr, sizeof(int) * 2048 * 1024, hipMemcpyHostToDevice);


	addcuda << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c); // ���� Block ������ Threads ����
	hipDeviceSynchronize();
	hipMemcpy(arr, d_c, sizeof(int) * 2048 * 1024, hipMemcpyDeviceToHost);

	for (int i = 0; i < 2048*1024; i++)
	{ 
		if (arr[i] != 1) printf(" NOT 1 \n");
	}

	hipFree(d_a); hipFree(d_b); hipFree(d_c);  free(arr);// 
}

int main()
{

	return 0;
}

*/

